#include "hip/hip_runtime.h"
/**
 * SPDX-FileCopyrightText: Copyright (c) 2021, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <morpheus/matx_functions.hpp>
#include <morpheus/type_utils.hpp>

#include <neo/cuda/sync.hpp>

#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <matx.h>

#include <memory>
#include <stdexcept>
#include <type_traits>

namespace morpheus {

struct matx_cast
{
    size_t element_count;
    rmm::cuda_stream_view stream;

    template <typename InputT,
              typename OutputT,
              std::enable_if_t<!cudf::is_numeric<InputT>() || !cudf::is_numeric<OutputT>()>* = nullptr>
    void operator()(void* input_data, void* output_data)
    {
        throw std::invalid_argument("Unsupported conversion");
    }

    template <typename InputT,
              typename OutputT,
              std::enable_if_t<cudf::is_numeric<InputT>() && cudf::is_numeric<OutputT>()>* = nullptr>
    void operator()(void* input_data, void* output_data)
    {
        matx::tensorShape_t<1> shape({static_cast<matx::index_t>(element_count)});

        matx::tensor_t<InputT, 1> input_tensor(static_cast<InputT*>(input_data), shape);
        matx::tensor_t<OutputT, 1> output_tensor(static_cast<OutputT*>(output_data), shape);

        (output_tensor = input_tensor).run(stream.value());
    }
};

std::shared_ptr<rmm::device_buffer> cast(const DevMemInfo& input, neo::TypeId output_type)
{
    auto input_dtype  = DType(input.type_id);
    auto output_dtype = DType(output_type);

    // Create the output
    auto output = std::make_shared<rmm::device_buffer>(
        output_dtype.item_size() * input.element_count, input.buffer->stream(), input.buffer->memory_resource());

    cudf::double_type_dispatcher(cudf::data_type{input_dtype.cudf_type_id()},
                                 cudf::data_type{output_dtype.cudf_type_id()},
                                 matx_cast{input.element_count, output->stream()},
                                 input.data(),
                                 output->data());

    neo::enqueue_stream_sync_event(output->stream()).get();

    return output;
}

struct matx_logits
{
    size_t element_count;
    rmm::cuda_stream_view stream;

    template <typename InputT, std::enable_if_t<!cudf::is_floating_point<InputT>()>* = nullptr>
    void operator()(void* input_data, void* output_data)
    {
        throw std::invalid_argument("Unsupported conversion");
    }

    template <typename InputT, std::enable_if_t<cudf::is_floating_point<InputT>()>* = nullptr>
    void operator()(void* input_data, void* output_data)
    {
        matx::tensorShape_t<1> shape({static_cast<matx::index_t>(element_count)});

        matx::tensor_t<InputT, 1> input_tensor(static_cast<InputT*>(input_data), shape);

        matx::tensor_t<InputT, 1> output_tensor(static_cast<InputT*>(output_data), shape);

        (output_tensor = (InputT)1 / ((InputT)1 + matx::exp((InputT)-1 * input_tensor))).run(stream.value());
    }
};

std::shared_ptr<rmm::device_buffer> logits(const DevMemInfo& input)
{
    auto input_dtype = DType(input.type_id);

    // Now create the output
    auto output = std::make_shared<rmm::device_buffer>(
        input_dtype.item_size() * input.element_count, input.buffer->stream(), input.buffer->memory_resource());

    cudf::type_dispatcher(cudf::data_type{input_dtype.cudf_type_id()},
                          matx_logits{input.element_count, output->stream()},
                          input.data(),
                          output->data());

    return output;
}

struct matx_transpose
{
    size_t element_count;
    rmm::cuda_stream_view stream;
    size_t rows;
    size_t cols;

    template <typename InputT, std::enable_if_t<!cudf::is_numeric<InputT>()>* = nullptr>
    void operator()(void* input_data, void* output_data)
    {
        throw std::invalid_argument("Unsupported conversion");
    }

    template <typename InputT, std::enable_if_t<cudf::is_numeric<InputT>()>* = nullptr>
    void operator()(void* input_data, void* output_data)
    {
        matx::tensorShape_t<2> input_shape({static_cast<matx::index_t>(rows), static_cast<matx::index_t>(cols)});
        matx::tensorShape_t<2> output_shape({static_cast<matx::index_t>(cols), static_cast<matx::index_t>(rows)});

        matx::tensor_t<InputT, 2> input_tensor(static_cast<InputT*>(input_data), input_shape);

        matx::tensor_t<InputT, 2> output_tensor(static_cast<InputT*>(output_data), output_shape);

        (output_tensor = input_tensor.Permute({1, 0})).run(stream.value());
    }
};

// Perform transpose
std::shared_ptr<rmm::device_buffer> transpose(const DevMemInfo& input, size_t rows, size_t cols)
{
    auto input_dtype = DType(input.type_id);

    // Now create the output
    auto output = std::make_shared<rmm::device_buffer>(
        input_dtype.item_size() * input.element_count, input.buffer->stream(), input.buffer->memory_resource());

    cudf::type_dispatcher(cudf::data_type{input_dtype.cudf_type_id()},
                          matx_transpose{input.element_count, output->stream(), rows, cols},
                          input.data(),
                          output->data());

    return output;
}



struct matx_create_seg_ids
{
    size_t element_count;
    size_t fea_len;
    rmm::cuda_stream_view stream;

    template <typename OutputT, std::enable_if_t<!std::is_integral_v<OutputT>>* = nullptr>
    void operator()(void* output_data)
    {
        throw std::invalid_argument("Unsupported conversion");
    }

    template <typename OutputT, std::enable_if_t<std::is_integral_v<OutputT>>* = nullptr>
    void operator()(void* output_data)
    {
        matx::tensorShape_t<2> shape({static_cast<matx::index_t>(element_count), 3});

        matx::tensor_t<OutputT, 2> output_tensor(static_cast<OutputT*>(output_data), shape);

        auto col0 = output_tensor.template Slice<1>({0, 0}, {matx::matxEnd, matx::matxDropDim});
        auto col2 = output_tensor.template Slice<1>({0, 2}, {matx::matxEnd, matx::matxDropDim});
        auto range_col =
            matx::range_x<OutputT>(matx::tensorShape_t<1>({static_cast<matx::index_t>(element_count)}), 0, 1);

        (col0 = range_col).run(stream.value());
        (col2 = fea_len - 1).run(stream.value());
    }
};

std::shared_ptr<rmm::device_buffer> create_seg_ids(size_t row_count, size_t fea_len, neo::TypeId output_type)
{
    auto output_dtype = DType(output_type);

    // Now create the output
    auto output =
        std::make_shared<rmm::device_buffer>(output_dtype.item_size() * row_count * 3, rmm::cuda_stream_per_thread);

    cudf::type_dispatcher(cudf::data_type{output_dtype.cudf_type_id()},
                          matx_create_seg_ids{row_count, fea_len, output->stream()},
                          output->data());

    return output;
}

}  // namespace morpheus
